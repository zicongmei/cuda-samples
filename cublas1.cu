#include <cstdio>
#include <cstdlib>
#include <vector>
#include <stdexcept>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


using data_type = double;

// CUDA API error checking
#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            std::printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__);                     \
            throw std::runtime_error("CUDA error");                                                \
        }                                                                                          \
    } while (0)

// cublas API error checking
#define CUBLAS_CHECK(err)                                                                          \
    do {                                                                                           \
        hipblasStatus_t err_ = (err);                                                               \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                                       \
            std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__);                   \
            throw std::runtime_error("cublas error");                                              \
        }                                                                                          \
    } while (0)

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    /*
     *   A = | 1.0 2.0 3.0 4.0 |
     */

    const std::vector<data_type> A = {1.0, 2.0, 3.0, 4.0};
    const int incx = 1;

    data_type result = 0.0;

    data_type *d_A = nullptr;

    printf("A\n");
    for (size_t i = 0; i< A.size(); ++i){
        printf("%f, ",A[i]);
    }
    printf("\n");
    //print_vector(A.size(), A.data());
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    CUBLAS_CHECK(hipblasDasum(cublasH, A.size(), d_A, incx, &result));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   result = 10.00
     */

    printf("result\n");
    std::printf("%0.2f\n", result);
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return 0;
}